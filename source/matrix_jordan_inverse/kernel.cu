#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#pragma comment(lib, "cuda.lib")
#pragma comment(lib, "cudart.lib")
#include <hip/hip_runtime.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include ""
#include <hipblas.h>

using namespace std;

__global__ void gaussjordan(float* A, float* I, int n, int i)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    float P;

    if (x < n && y < n)
        if (x > i) { // this limits operation to rows below the pivot point
            P = A[x * n + i] / A[i * n + i];
            I[x * n + y] -= I[i * n + y] * P;  // apply for every row member
            if (y >= i) { //limits  to row members to the right of the pivot
                A[x * n + y] -= A[i * n + y] * P;  // apply only to members right of pivot
            }
        }
}


__global__ void dev(float* d_A, float* dI, int h)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < h && y < h)
        if (d_A[x * h + x] != 0) {
            dI[x * h + y] /= d_A[x * h + x];
            d_A[x * h + y] /= d_A[x * h + x];
        }
    __syncthreads();

}

void savetofile(float* A, string s, int n, int h)
{
    std::ofstream plik;
    plik.open(s);

    for (int j = 0; j < h; j++) {
        for (int i = 0; i < h; i++) {
            plik << A[j * n + i] << "\t";
        }
        plik << endl;
    }
    plik.close();
}

void random_floats(float* vect, int N) {
    for (int i = 0; i < N; i++) {
        vect[i] =static_cast<float>(rand())/(static_cast<float>(RAND_MAX/50));
    }
}


int main()
{
    int n = 16;
    // creating input
    float* iL = new float[n * n];
    float* L = new float[n * n];
    random_floats(L, n * n);
    savetofile(L, "Input_matrix.txt", n, n);

    cout << "inv\n";
    float* d_A, * I, * dI;
    float time;
    hipError_t err;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    int ddsize = n * n * sizeof(float);

    dim3 threadsPerBlock(n / 16, n / 16);
    dim3 numBlocks(16, 16);
    // memory allocation    
    err = hipMalloc((void**)&d_A, ddsize);   if (err != hipSuccess) { cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << endl; }
    err = hipMalloc((void**)&dI, ddsize);   if (err != hipSuccess) { cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << endl; }
    I = new float[n * n];

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            if (i == j) I[i * n + i] = 1.0;
            else I[i * n + j] = 0.0;
        }
    }
    //copy data from GPU to CPU
    err = hipMemcpy(d_A, L, ddsize, hipMemcpyHostToDevice); if (err != hipSuccess) { cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << endl; }
    err = hipMemcpy(dI, I, ddsize, hipMemcpyHostToDevice);  if (err != hipSuccess) { cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << endl; }
    //timer start
    hipEventRecord(start, 0);
    // Calculating inverse using Gauss-Jordan elimination
    for (int i = 0; i < n; i++) {
        gaussjordan << <numBlocks, threadsPerBlock >> > (d_A, dI, n, i);
    }
    dev << <numBlocks, threadsPerBlock >> > (d_A, dI, n);

    err = hipMemcpy(iL, dI, ddsize, hipMemcpyDeviceToHost); if (err != hipSuccess) { cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << endl; }
    err = hipMemcpy(L, d_A, ddsize, hipMemcpyDeviceToHost); if (err != hipSuccess) { cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << endl; }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    std::cout << "Cuda Time - inverse: " << time << "ms\n";
    savetofile(iL, "inverse.txt", n, n);
    hipFree(d_A);
    hipFree(dI);
    delete[]I;
    delete[]L;
    delete[]iL;
    system("Pause");
    return 0;
}