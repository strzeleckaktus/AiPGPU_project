﻿#include "hip/hip_runtime.h"


#include <iostream>
#include <cstdlib>
#include <ctime>

using namespace std;

#define BLOCK_SIZE 32
#define MULTIPLICATOR 5

__global__ void Matrix_multiplication(float* A, float* B, float mul, int width)
{
    B[blockIdx.x * blockDim.x + threadIdx.x] = A[blockIdx.x * blockDim.x + threadIdx.x] * mul;
}

void random_ints(float** matrix, size_t N, size_t M) {
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < M; j++) {
            matrix[i][j] = (float)(rand() % 10);
        }
    }
}

void printResults(float** A, float** B, int width) {
    printf("Matrix A:\n");
    for (int i = 0; i < width; i++) {
        for (int j = 0; j < width; j++) {
            printf("%f	", A[i][j]);
        }
        printf("\n");
    }
    printf("Matrix B:\n");
    for (int i = 0; i < width; i++) {
        for (int j = 0; j < width; j++) {
            printf("%f	", B[i][j]);
        }
        printf("\n");
    }
}

int main() {
    int N = 4;

    float** A = new float* [N];
    A[0] = new float[N * N];
    for (int i = 1; i < N; i++) {
        A[i] = A[0] + i * N;
    }
    random_ints(A, N, N);

    float** B = new float* [N];
    B[0] = new float[N * N];
    for (int i = 1; i < N; i++) {
        B[i] = B[0] + i * N;
    }

    float* cuda_A, * cuda_B;

    hipMalloc(&cuda_A, (N * N) * sizeof(float));
    hipMalloc(&cuda_B, (N * N) * sizeof(float));

    hipMemcpy(cuda_A, A[0], (N * N) * sizeof(float), hipMemcpyHostToDevice);

    Matrix_multiplication <<< ((N * N) + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE >>> (cuda_A, cuda_B, MULTIPLICATOR, N);

    hipMemcpy(B[0], cuda_B, (N * N) * sizeof(float), hipMemcpyDeviceToHost);

    printResults(A, B, N);

    hipFree(cuda_A);
    hipFree(cuda_B);
    delete(A);
    delete(B);
    system("pause");
    return 0;

}