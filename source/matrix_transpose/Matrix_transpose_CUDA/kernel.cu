#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <cstdlib>
#include <ctime>

using namespace std;

#define TILE_DIM 3

__global__ void Matrix_transpose(float* A, float* B, int A_rows, int A_cols) {

    float CValue = 0;

    int Row = blockIdx.y * TILE_DIM + threadIdx.y;
    int Col = blockIdx.x * TILE_DIM + threadIdx.x;

    __shared__ float As[TILE_DIM][TILE_DIM];

    if (Row < A_rows && Col < A_cols)
        As[threadIdx.x][threadIdx.y] = A[Row * A_cols + Col];

    //B[Row + Col*A_cols] = As[threadIdx.x][threadIdx.y];
    if (Row < A_rows && Col < A_cols)
        B[Row + Col * A_cols] = As[threadIdx.x][threadIdx.y];
}

void random_ints(float** matrix, size_t N, size_t M) {
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < M; j++) {
            matrix[i][j] = (float)i*N + j;
        }
    }
}

void printResults(float** A, float** B, size_t N, size_t M) {
    printf("Matrix A:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < M; j++) {
            printf("%f	", A[i][j]);
        }
        printf("\n");
    }
    printf("Matrix B:\n");
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            printf("%f	", B[i][j]);
        }
        printf("\n");
    }
}

int main() {
    int N = 5;
    int M = 5;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float** A = new float* [N];
    A[0] = new float[M * N];
    for (int i = 1; i < N; i++) {
        A[i] = A[0] + i * M;
    }
    random_ints(A, N, M);

    float** B = new float* [M];
    B[0] = new float[M * N];
    for (int i = 1; i < M; i++) {
        B[i] = B[0] + i * N;
    }

    float* cuda_A, * cuda_B;

    hipMalloc(&cuda_A, (N * M) * sizeof(float));
    hipMalloc(&cuda_B, (N * M) * sizeof(float));

    hipMemcpy(cuda_A, A[0], (N * M) * sizeof(float), hipMemcpyHostToDevice);

    dim3 dimGrid((N * TILE_DIM - 1) / TILE_DIM, (N * TILE_DIM - 1) / TILE_DIM);
    dim3 dimBlock(TILE_DIM, TILE_DIM);

    hipEventRecord(start);

    Matrix_transpose << <dimGrid, dimBlock >> > (cuda_A, cuda_B, N, M);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipMemcpy(B[0], cuda_B, (N * M) * sizeof(float), hipMemcpyDeviceToHost);

    printResults(A, B, N, M);
    //printResults(A, B, N, M);
    hipFree(cuda_A);
    hipFree(cuda_B);
    delete(A);
    delete(B);

    system("pause");

    return 0;
}